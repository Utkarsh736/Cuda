#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256  // Number of threads per block

// CUDA Kernel for ReLU Activation
__global__ void relu_kernel(float *input, float *output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = fmaxf(0.0f, input[idx]);  // ReLU: max(0, x)
    }
}

// Host function to launch the CUDA kernel
void relu(float *h_input, float *h_output, int n) {
    float *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    int grid_size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch ReLU kernel
    relu_kernel<<<grid_size, BLOCK_SIZE>>>(d_input, d_output, n);

    // Copy result back to host
    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

// Helper function to print an array
void printArray(float *arr, int n) {
    for (int i = 0; i < n; i++) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

// Main function
int main() {
    const int n = 10;
    float h_input[n]  = {-3.0, -2.0, -1.0, 0.0, 1.0, 2.0, 3.0, 4.0, -5.0, 6.0};
    float h_output[n];

    std::cout << "Input:  ";
    printArray(h_input, n);

    // Compute ReLU
    relu(h_input, h_output, n);

    std::cout << "ReLU Output: ";
    printArray(h_output, n);

    return 0;
}
