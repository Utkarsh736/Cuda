#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4       // Number of rows in matrix
#define NNZ 7     // Number of non-zero elements
#define BLOCK_SIZE 4  // Set block size to match row count for simplicity

// Kernel for SpMV using CSR with shared memory optimization
__global__ void spmv_csr_shared(int *rowPtr, int *colIdx, float *values, float *x, float *y, int rows) {
    __shared__ float x_shared[N];  // Shared memory for input vector

    int thread_id = threadIdx.x;
    int row = blockIdx.x * blockDim.x + thread_id;

    // Load input vector `x` into shared memory
    if (thread_id < N) {
        x_shared[thread_id] = x[thread_id];
    }
    __syncthreads();

    if (row < rows) {
        float sum = 0.0;
        for (int i = rowPtr[row]; i < rowPtr[row + 1]; i++) {
            sum += values[i] * x_shared[colIdx[i]];
        }
        y[row] = sum;
    }
}

int main() {
    // Host data: CSR representation of a sparse matrix
    int h_rowPtr[N+1] = {0, 2, 4, 5, 7};  // Row pointers
    int h_colIdx[NNZ] = {0, 1, 1, 2, 2, 0, 3};  // Column indices
    float h_values[NNZ] = {1, 2, 3, 4, 5, 6, 7};  // Non-zero values
    float h_x[N] = {1, 2, 3, 4};  // Input vector
    float h_y[N] = {0, 0, 0, 0};  // Output vector

    // Device memory allocation
    int *d_rowPtr, *d_colIdx;
    float *d_values, *d_x, *d_y;
    hipMalloc(&d_rowPtr, (N+1) * sizeof(int));
    hipMalloc(&d_colIdx, NNZ * sizeof(int));
    hipMalloc(&d_values, NNZ * sizeof(float));
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_rowPtr, h_rowPtr, (N+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, h_colIdx, NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch with shared memory
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    spmv_csr_shared<<<blocksPerGrid, threadsPerBlock>>>(d_rowPtr, d_colIdx, d_values, d_x, d_y, N);

    // Copy result back to host
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Result (y = A * x):\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_y[i]);
    }

    // Free memory
    hipFree(d_rowPtr);
    hipFree(d_colIdx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
