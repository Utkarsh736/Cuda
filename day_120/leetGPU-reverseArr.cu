#include <hip/hip_runtime.h>
#include <iostream>

__global__ void reverseKernel(float* input, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int mirror_idx = N - 1 - idx;

    if (idx < N / 2) {
        float temp = input[idx];
        input[idx] = input[mirror_idx];
        input[mirror_idx] = temp;
    }
}

void solve(float* input, int N) {
    float* d_input;
    size_t size = N * sizeof(float);

    hipMalloc((void**)&d_input, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N / 2 + threadsPerBlock - 1) / threadsPerBlock;

    reverseKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, N);

    hipMemcpy(input, d_input, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
}
