#include <hip/hip_runtime.h>
#include <iostream>

__global__ void leakyReluKernel(const float* input, float* output, int N, float alpha) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = input[idx];
        output[idx] = (x >= 0.0f) ? x : alpha * x;
    }
}

void solve(const float* input, float* output, int N) {
    const float alpha = 0.01f;
    float *d_input, *d_output;
    size_t size = N * sizeof(float);

    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    leakyReluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N, alpha);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
