// mse_loss.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mse_kernel(const float* preds, const float* targets, float* partial_sums, int N) {
    __shared__ float cache[256];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    float temp = 0.0f;

    if (i < N) {
        float diff = preds[i] - targets[i];
        temp = diff * diff;
    }

    cache[tid] = temp;
    __syncthreads();

    // Reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s)
            cache[tid] += cache[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        partial_sums[blockIdx.x] = cache[0];
}

extern "C" void solve(const float* h_predictions, const float* h_targets, int N, float* h_mse) {
    float *d_predictions, *d_targets, *d_partial, *h_partial;
    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    hipMalloc(&d_predictions, N * sizeof(float));
    hipMalloc(&d_targets, N * sizeof(float));
    hipMalloc(&d_partial, blocks * sizeof(float));
    h_partial = (float*)malloc(blocks * sizeof(float));

    hipMemcpy(d_predictions, h_predictions, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_targets, h_targets, N * sizeof(float), hipMemcpyHostToDevice);

    mse_kernel<<<blocks, threads>>>(d_predictions, d_targets, d_partial, N);
    hipMemcpy(h_partial, d_partial, blocks * sizeof(float), hipMemcpyDeviceToHost);

    float total = 0.0f;
    for (int i = 0; i < blocks; ++i)
        total += h_partial[i];

    *h_mse = total / N;

    hipFree(d_predictions);
    hipFree(d_targets);
    hipFree(d_partial);
    free(h_partial);
}
