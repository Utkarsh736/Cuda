#include "hip/hip_runtime.h"
//!POPCORN leaderboard amd-identity

#include <hip/hip_runtime.h>

// Device kernel: copies each element from input to output
extern "C" __global__ void identity_kernel(const float* __restrict__ input,
                                           float* __restrict__ output,
                                           size_t N) {
    size_t idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    if (idx < N) {
        output[idx] = input[idx];
    }
}

// Host wrapper called by the evaluation harness
void custom_kernel(const torch::Tensor& input_tensor,
                   torch::Tensor& output_tensor) {
    const float* input  = input_tensor.data_ptr<float>();
    float*       output = output_tensor.data_ptr<float>();
    size_t       N      = input_tensor.numel();

    // Launch parameters
    constexpr size_t THREADS = 256;
    size_t num_blocks = (N + THREADS - 1) / THREADS;

    // Launch the HIP kernel 0
    hipLaunchKernelGGL(identity_kernel,
                       dim3(num_blocks), dim3(THREADS),
                       0, 0,
                       input, output, N);

    // Wait for completion before returning 1
    hipDeviceSynchronize();
}