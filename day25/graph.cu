#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define INF 2147483647  // Representation of infinity

__global__ void bfs_kernel(int* d_nodes, int* d_edges, int* d_cost, int* d_frontier, int* d_next_frontier, int* d_frontier_size, int* d_next_frontier_size, int level) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < *d_frontier_size) {
        int node = d_frontier[tid];
        int start = d_nodes[node];
        int end = d_nodes[node + 1];
        for (int i = start; i < end; i++) {
            int neighbor = d_edges[i];
            if (d_cost[neighbor] == INF) {
                d_cost[neighbor] = level + 1;
                int index = atomicAdd(d_next_frontier_size, 1);
                d_next_frontier[index] = neighbor;
            }
        }
    }
}

void bfs(int* h_nodes, int* h_edges, int num_nodes, int source) {
    // Allocate host memory
    int* h_cost = (int*)malloc(num_nodes * sizeof(int));
    int* h_frontier = (int*)malloc(num_nodes * sizeof(int));
    int* h_next_frontier = (int*)malloc(num_nodes * sizeof(int));
    int h_frontier_size = 1;
    int h_next_frontier_size = 0;

    // Initialize host memory
    for (int i = 0; i < num_nodes; i++) {
        h_cost[i] = INF;
    }
    h_cost[source] = 0;
    h_frontier[0] = source;

    // Allocate device memory
    int *d_nodes, *d_edges, *d_cost, *d_frontier, *d_next_frontier, *d_frontier_size, *d_next_frontier_size;
    hipMalloc((void**)&d_nodes, (num_nodes + 1) * sizeof(int));
    hipMalloc((void**)&d_edges, h_nodes[num_nodes] * sizeof(int));
    hipMalloc((void**)&d_cost, num_nodes * sizeof(int));
    hipMalloc((void**)&d_frontier, num_nodes * sizeof(int));
    hipMalloc((void**)&d_next_frontier, num_nodes * sizeof(int));
    hipMalloc((void**)&d_frontier_size, sizeof(int));
    hipMalloc((void**)&d_next_frontier_size, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_nodes, h_nodes, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, h_edges, h_nodes[num_nodes] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cost, h_cost, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier, h_frontier, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_size, &h_frontier_size, sizeof(int), hipMemcpyHostToDevice);

    int level = 0;
    while (h_frontier_size > 0) {
        int threads_per_block = 256;
        int blocks_per_grid = (h_frontier_size + threads_per_block - 1) / threads_per_block;

        // Initialize next frontier size on device
        hipMemcpy(d_next_frontier_size, &h_next_frontier_size, sizeof(int), hipMemcpyHostToDevice);

        // Launch BFS kernel
        bfs_kernel<<<blocks_per_grid, threads_per_block>>>(d_nodes, d_edges, d_cost, d_frontier, d_next_frontier, d_frontier_size, d_next_frontier_size, level);
        hipDeviceSynchronize();

        // Copy next frontier size from device to host
        hipMemcpy(&h_next_frontier_size, d_next_frontier_size, sizeof(int), hipMemcpyDeviceToHost);

        // Swap frontiers
        int* temp = d_frontier;
        d_frontier = d_next_frontier;
        d_next_frontier = temp;

        h_frontier_size = h_next_frontier_size;
        h_next_frontier_size = 0;
        level++;

        // Copy new frontier size to device
        hipMemcpy(d_frontier_size, &h_frontier_size, sizeof(int), hipMemcpyHostToDevice);
    }

    // Copy result from device to host
    hipMemcpy(h_cost, d_cost, num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    // Output the cost array
    for (int i = 0; i < num_nodes; i++) {
        if (h_cost[i] == INF) {
            printf("Node %d is unreachable\n", i);
        } else {
            printf("Cost to reach node %d is %d\n", i, h_cost[i]);
        }
    }

    // Free device memory
    hipFree(d_nodes);
    hipFree(d_edges);
    hipFree(d_cost);
    hipFree(d_frontier);
    hipFree(d_next_frontier);
    hipFree(d_frontier_size);
    hipFree(d_next_frontier_size);

    // Free host memory
    free(h_cost);
    free(h_frontier);
    free(h_next_frontier);
}

int main() {
    // Example graph in CSR format
    int h_nodes[] = {0, 2, 5, 7, 9};  // Node pointers
    int h_edges[] = {1, 2, 0, 2, 3, 0, 1, 1, 2};  // Edges
    int num_nodes = 4;
    int source = 0;

    bfs(h_nodes, h_edges, num_nodes, source);

    return 0;
}
